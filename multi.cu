
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 512 // Matrix size

__global__ void matrixMul(int *a, int *b, int *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        int sum = 0;
        for (int i = 0; i < N; ++i) {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

int main() {
    int *a, *b, *c; // Host matrices
    int *d_a, *d_b, *d_c; // Device matrices

    // Allocate memory on host
    a = (int*)malloc(N * N * sizeof(int));
    b = (int*)malloc(N * N * sizeof(int));
    c = (int*)malloc(N * N * sizeof(int));

    // Allocate memory on device
    hipMalloc(&d_a, N * N * sizeof(int));
    hipMalloc(&d_b, N * N * sizeof(int));
    hipMalloc(&d_c, N * N * sizeof(int));

    // Initialize matrices on host
    for (int i = 0; i < N * N; ++i) {
        a[i] = i;
        b[i] = i;
    }

    // Copy matrices from host to device
    hipMemcpy(d_a, a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    matrixMul<<<gridSize, blockSize>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify results
    printf("First element of result matrix: %d\n", c[0]);

    // Free memory
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}

